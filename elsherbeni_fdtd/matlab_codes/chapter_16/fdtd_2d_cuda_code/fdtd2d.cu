#include "hip/hip_runtime.h"
 
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "fdtd2d_kernel.cu"
#include "global.h"

texture<float4, 2, hipReadModeElementType> inTex;

__constant__ unsigned int  dvrgb[256];

__global__ void 
find_min_and_max_on_gpu(int nblocks, float* field, float* minimum_field_value, float* maximum_field_value)
{
	__shared__ float minarr[512];
	__shared__ float maxarr[512];
	
	int i = threadIdx.x;
	int nTotalThreads = blockDim.x;

	minarr[i] = field[i];
	maxarr[i] = minarr[i];
	for (int j=1;j<nblocks;j++)
	{
		minarr[i+nTotalThreads] = field[i+nTotalThreads*j];
		if (minarr[i] > minarr[i+nTotalThreads])
				minarr[i] = minarr[i+nTotalThreads];

		if (maxarr[i] < minarr[i+nTotalThreads])
				maxarr[i] = minarr[i+nTotalThreads];
		__syncthreads();
	}
		__syncthreads();

	while(nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		if (threadIdx.x < halfPoint)
		{
			float temp = minarr[i + halfPoint];

			if (temp < minarr[i]) minarr[i] = temp;

			temp = maxarr[i + halfPoint];
			if (temp > maxarr[i]) maxarr[i] = temp;
		}
		__syncthreads();
		nTotalThreads = (nTotalThreads >> 1);	
	}
	if (i == 0)
	{
		minimum_field_value[0] = minarr[0];
		maximum_field_value[0] = maxarr[0];
	}
}

void createColormapOnGpu()
{
	hipError_t et;	

unsigned int rgb[] = {4286775296,4287037440,4287299584,4287561728,4287823872,4288086016,4288348160,
4288610304,4288872448,4289134592,4289396736,4289658880,4289921024,4290183168,
4290445312,4290707456,4290969600,4291231744,4291493888,4291756032,4292018176,
4292280320,4292542464,4292804608,4293066752,4293328896,4293591040,4293853184,
4294115328,4294377472,4294639616,4294901760,4294902784,4294903808,4294904832,
4294905856,4294906880,4294907904,4294908928,4294909952,4294910976,4294912000,
4294913024,4294914048,4294915072,4294916096,4294917120,4294918144,4294919168,
4294920192,4294921216,4294922240,4294923264,4294924288,4294925312,4294926336,
4294927360,4294928384,4294929408,4294930432,4294931456,4294932480,4294933504,
4294934528,4294935296,4294936320,4294937344,4294938368,4294939392,4294940416,
4294941440,4294942464,4294943488,4294944512,4294945536,4294946560,4294947584,
4294948608,4294949632,4294950656,4294951680,4294952704,4294953728,4294954752,
4294955776,4294956800,4294957824,4294958848,4294959872,4294960896,4294961920,
4294962944,4294963968,4294964992,4294966016,4294967040,4294704900,4294442760,
4294180620,4293918480,4293656340,4293394200,4293132060,4292869920,4292607780,
4292345640,4292083500,4291821360,4291559220,4291297080,4291034940,4290772800,
4290510660,4290248520,4289986380,4289724240,4289462100,4289199960,4288937820,
4288675680,4288413540,4288151400,4287889260,4287627120,4287364980,4287102840,
4286840700,4286644096,4286381955,4286119815,4285857675,4285595535,4285333395,
4285071255,4284809115,4284546975,4284284835,4284022695,4283760555,4283498415,
4283236275,4282974135,4282711995,4282449855,4282187715,4281925575,4281663435,
4281401295,4281139155,4280877015,4280614875,4280352735,4280090595,4279828455,
4279566315,4279304175,4279042035,4278779895,4278517755,4278255615,4278254591,
4278253567,4278252543,4278251519,4278250495,4278249471,4278248447,4278247423,
4278246399,4278245375,4278244351,4278243327,4278242303,4278241279,4278240255,
4278239231,4278238207,4278237183,4278236159,4278235135,4278234111,4278233087,
4278232063,4278231039,4278230015,4278228991,4278227967,4278226943,4278225919,
4278224895,4278223871,4278223103,4278222079,4278221055,4278220031,4278219007,
4278217983,4278216959,4278215935,4278214911,4278213887,4278212863,4278211839,
4278210815,4278209791,4278208767,4278207743,4278206719,4278205695,4278204671,
4278203647,4278202623,4278201599,4278200575,4278199551,4278198527,4278197503,
4278196479,4278195455,4278194431,4278193407,4278192383,4278191359,4278190335,
4278190331,4278190327,4278190323,4278190319,4278190315,4278190311,4278190307,
4278190303,4278190299,4278190295,4278190291,4278190287,4278190283,4278190279,
4278190275,4278190271,4278190267,4278190263,4278190259,4278190255,4278190251,
4278190247,4278190243,4278190239,4278190235,4278190231,4278190227,4278190223,
4278190219,4278190215,4278190211,4278190208};

	et = hipMemcpyToSymbol(HIP_SYMBOL( dvrgb), rgb, 256*sizeof(int), 0, hipMemcpyHostToDevice); 
}

__global__ void 
create_image_on_gpu(unsigned int* g_odata, float* Ez, int nxx, float minval, float maxval)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int cind;
	float F;

	int ci = j*nxx+i;
	int ti = (j+1)*nxx+i;
	F = Ez[ti] - minval;
	cind = floor(255 * F/(maxval-minval));
	if (cind > 255) cind = 255;
	g_odata [ci] = dvrgb[cind];
}

extern "C" void
createImageOnGpu(unsigned int* g_odata)
{
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid(nxx/block.x, nyy/block.y, 1);
	dim3 gridm  = dim3(1,1,1);
	dim3 blockm = dim3(TILE_SIZE*TILE_SIZE,1,1);
	int  nblocks = grid.x * grid.y;
	float minval;
	float maxval;
	float *dvF;

	if (show_Ez) dvF=dvEz; else dvF=dvHz;

	find_min_and_max_on_gpu<<< gridm, blockm>>>(nblocks, dvF, dvminimum_field_value, dvmaximum_field_value);

	hipMemcpy(&minval, dvminimum_field_value, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&maxval, dvmaximum_field_value, sizeof(float), hipMemcpyDeviceToHost);
	
	if (minval>0.0) minval = 0.0;
	if (maxval<0.0) maxval = 0.0;
	if (abs(minval)>maxval) maxval = -minval; else minval = -maxval;
	if (minval<global_min_field) global_min_field = minval;
	if (maxval>global_max_field) global_max_field = maxval;

	create_image_on_gpu<<< grid, block>>>(g_odata, dvF, nxx, global_min_field, global_max_field);
}

bool check_device()
{
	int currentDevice;
    hipDeviceProp_t deviceProp;
	int multiProcessorCount;
	int compute_capability_major;
	int compute_capability_minor;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if( deviceCount == 0 )
    {
        printf("No gpu devices found!\n");
        return false;
    } 

	if (hipGetDevice (&currentDevice) != hipSuccess) 
	{
        printf("No gpu devices available!\n");
        return false;
	}

	if (hipGetDeviceProperties(&deviceProp, currentDevice) != hipSuccess) 
	{
        printf("No gpu devices available!\n");
        return false;
	}

	maximum_threads_per_block  = deviceProp.maxThreadsPerBlock;
	multiProcessorCount = deviceProp.multiProcessorCount;
	compute_capability_major = deviceProp.major;
	compute_capability_minor = deviceProp.minor;

	printf("Current Device %d: %s\n", currentDevice, deviceProp.name);
	printf("    Multi Processor Count: %d\n", multiProcessorCount);
	printf("    Maximum Threads per Block: %d\n", maximum_threads_per_block);
	printf("    Compute Capability: %d.%d \n", compute_capability_major, compute_capability_minor);

	return true;
}


int copyFdtdArraysToGpuMemory()
{
	int size_int   = sizeof(int);
	int size_char  = sizeof(char);
	int size_float = sizeof(float);
	int array_size  = number_of_cells * size_float; 
	int array_size_with_pads  = number_of_cells_with_pads * size_float; 

	hipError_t et;	

	et = hipMalloc((void**)&dvEx, array_size_with_pads);     if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvEy, array_size_with_pads);     if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvEz, array_size_with_pads);     if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvHx, array_size_with_pads);     if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvHy, array_size_with_pads);     if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvHz, array_size_with_pads);     if (et == hipErrorOutOfMemory) return 1;

	et = hipMalloc((void**)&dvCexe,  array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvCexhz, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvCeye,  array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvCeyhz, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvCeze,  array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvCezhy, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvCezhx, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;

	et = hipMalloc((void**)&dvChxh,  array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvChxez, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvChyh,  array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvChyez, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvChzh,  array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvChzex, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;
	et = hipMalloc((void**)&dvChzey, array_size_with_pads);    if (et == hipErrorOutOfMemory) return 1;

	hipMemcpy(dvEx, Ex, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvEy, Ey, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvEz, Ez, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvHx, Hx, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvHy, Hy, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvHz, Hz, array_size_with_pads, hipMemcpyHostToDevice);

	hipMemcpy(dvCexe,  Cexe,  array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvCexhz, Cexhz, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvCeye,  Ceye,  array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvCeyhz, Ceyhz, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvCeze,  Ceze,  array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvCezhy, Cezhy, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvCezhx, Cezhx, array_size_with_pads, hipMemcpyHostToDevice);

	hipMemcpy(dvChxh,  Chxh,  array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvChxez, Chxez, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvChyh,  Chyh,  array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvChyez, Chyez, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvChzh,  Chzh,  array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvChzex, Chzex, array_size_with_pads, hipMemcpyHostToDevice);
	hipMemcpy(dvChzey, Chzey, array_size_with_pads, hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvJ_indices, total_number_of_cej_components*sizeof(int));     if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvJ_indices,  J_indices, total_number_of_cej_components*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvimpressed_J_Cej, total_number_of_cej_components*sizeof(int));     if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvimpressed_J_Cej,  impressed_J_Cej, total_number_of_cej_components*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvM_indices, total_number_of_chm_components*sizeof(int));     if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvM_indices,  M_indices, total_number_of_chm_components*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvimpressed_M_Chm, total_number_of_chm_components*sizeof(int));     if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvimpressed_M_Chm,  impressed_M_Chm, total_number_of_chm_components*sizeof(int), hipMemcpyHostToDevice);


	et = hipMalloc((void**)&dvsampled_magnetic_fields_component, number_of_sampled_magnetic_fields*sizeof(char));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_magnetic_fields_component,  sampled_magnetic_fields_component, number_of_sampled_magnetic_fields*sizeof(char), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvsampled_magnetic_fields_is, number_of_sampled_magnetic_fields*sizeof(int));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_magnetic_fields_is,  sampled_magnetic_fields_is, number_of_sampled_magnetic_fields*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvsampled_magnetic_fields_js, number_of_sampled_magnetic_fields*sizeof(int));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_magnetic_fields_js,  sampled_magnetic_fields_js, number_of_sampled_magnetic_fields*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvsampled_magnetic_fields_sampled_value, number_of_sampled_magnetic_fields*number_of_time_steps*sizeof(float));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_magnetic_fields_sampled_value, sampled_magnetic_fields_sampled_value, number_of_sampled_magnetic_fields*number_of_time_steps*sizeof(float), hipMemcpyHostToDevice);


	et = hipMalloc((void**)&dvsampled_electric_fields_component, number_of_sampled_electric_fields*sizeof(char));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_electric_fields_component,  sampled_electric_fields_component, number_of_sampled_electric_fields*sizeof(char), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvsampled_electric_fields_is, number_of_sampled_electric_fields*sizeof(int));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_electric_fields_is,  sampled_electric_fields_is, number_of_sampled_electric_fields*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvsampled_electric_fields_js, number_of_sampled_electric_fields*sizeof(int));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_electric_fields_js,  sampled_electric_fields_js, number_of_sampled_electric_fields*sizeof(int), hipMemcpyHostToDevice);

	et = hipMalloc((void**)&dvsampled_electric_fields_sampled_value, number_of_sampled_electric_fields*number_of_time_steps*sizeof(float));     
	if (et == hipErrorOutOfMemory) return 1;
	hipMemcpy(dvsampled_electric_fields_sampled_value, sampled_electric_fields_sampled_value, number_of_sampled_electric_fields*number_of_time_steps*sizeof(float), hipMemcpyHostToDevice);

	if (is_cpml_xn)
	{
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_ex_xn), cpml_a_ex_xn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_ex_xn), cpml_b_ex_xn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_mx_xn), cpml_a_mx_xn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_mx_xn), cpml_b_mx_xn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 

		if (is_TEz)
		{
			et = hipMalloc((void**)&dvPsi_hzx_xn,  cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hzx_xn , Psi_hzx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hzx_xn, cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hzx_xn , CPsi_hzx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_eyx_xn,  cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_eyx_xn , Psi_eyx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_eyx_xn, cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_eyx_xn , CPsi_eyx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
		}
		if (is_TMz)
		{
			et = hipMalloc((void**)&dvPsi_ezx_xn,  cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_ezx_xn , Psi_ezx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_ezx_xn, cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_ezx_xn , CPsi_ezx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_hyx_xn,  cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hyx_xn , Psi_hyx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hyx_xn, cpml_2d_array_size_xn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hyx_xn , CPsi_hyx_xn, cpml_2d_array_size_xn, hipMemcpyHostToDevice);
		}
	}

	if (is_cpml_xp)
	{
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_ex_xp), cpml_a_ex_xp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_ex_xp), cpml_b_ex_xp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_mx_xp), cpml_a_mx_xp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_mx_xp), cpml_b_mx_xp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 

		if (is_TEz)
		{
			et = hipMalloc((void**)&dvPsi_hzx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hzx_xp , Psi_hzx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hzx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hzx_xp , CPsi_hzx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_eyx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_eyx_xp , Psi_eyx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_eyx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_eyx_xp , CPsi_eyx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
		}
		if (is_TMz)
		{
			et = hipMalloc((void**)&dvPsi_ezx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_ezx_xp , Psi_ezx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_ezx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_ezx_xp , CPsi_ezx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_hyx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hyx_xp , Psi_hyx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hyx_xp, cpml_2d_array_size_xp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hyx_xp , CPsi_hyx_xp, cpml_2d_array_size_xp, hipMemcpyHostToDevice);
		}
	}

	if (is_cpml_yn)
	{
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_ey_yn), cpml_a_ey_yn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_ey_yn), cpml_b_ey_yn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_my_yn), cpml_a_my_yn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_my_yn), cpml_b_my_yn, cpml_1d_array_size_xnyn, 0, hipMemcpyHostToDevice); 
		if (is_TEz)
		{
			et = hipMalloc((void**)&dvPsi_hzy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hzy_yn , Psi_hzy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hzy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hzy_yn , CPsi_hzy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_exy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_exy_yn , Psi_exy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_exy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_exy_yn , CPsi_exy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
		}
		if (is_TMz)
		{
			et = hipMalloc((void**)&dvPsi_ezy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_ezy_yn , Psi_ezy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_ezy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_ezy_yn , CPsi_ezy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_hxy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hxy_yn , Psi_hxy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hxy_yn, cpml_2d_array_size_yn);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hxy_yn , CPsi_hxy_yn, cpml_2d_array_size_yn, hipMemcpyHostToDevice);
		}
	}

	if (is_cpml_yp)
	{
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_ey_yp), cpml_a_ey_yp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_ey_yp), cpml_b_ey_yp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_a_my_yp), cpml_a_my_yp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		et = hipMemcpyToSymbol(HIP_SYMBOL(dvcpml_b_my_yp), cpml_b_my_yp, cpml_1d_array_size_xpyp, 0, hipMemcpyHostToDevice); 
		if (is_TEz)
		{
			et = hipMalloc((void**)&dvPsi_hzy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hzy_yp , Psi_hzy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hzy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hzy_yp , CPsi_hzy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_exy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_exy_yp , Psi_exy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_exy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_exy_yp , CPsi_exy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
		}
		if (is_TMz)
		{
			et = hipMalloc((void**)&dvPsi_ezy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_ezy_yp , Psi_ezy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_ezy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_ezy_yp , CPsi_ezy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvPsi_hxy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_hxy_yp , Psi_hxy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
			et = hipMalloc((void**)&dvCPsi_hxy_yp, cpml_2d_array_size_yp);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvCPsi_hxy_yp , CPsi_hxy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
		}
	}

	et = hipMalloc((void**)&dvminimum_field_value, sizeof(float)*TILE_SIZE);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_ezy_yp , Psi_ezy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);
	et = hipMalloc((void**)&dvmaximum_field_value, sizeof(float)*TILE_SIZE);	if (et == hipErrorOutOfMemory) return 1;	hipMemcpy(dvPsi_ezy_yp , Psi_ezy_yp, cpml_2d_array_size_yp, hipMemcpyHostToDevice);

	return 0;
}

void update_impressed_magnetic_currents_on_gpu(int time_step)
{
	dim3 threads_M, grid_M;
	float *dvH;

	int chm_ind = 0;
	for (int i=0;i<number_of_impressed_M;i++)
	{
		threads_M = dim3(number_of_chm_components[i], 1, 1);
		grid_M    = dim3(1, 1, 1);
		
		if (impressed_M_direction[i]=='x') dvH = dvHx;
		else if (impressed_M_direction[i]=='y')	dvH = dvHy; else dvH = dvHz;

		update_impressed_M_on_gpu<<<grid_M, threads_M>>>
			(dvH,  dvM_indices, dvimpressed_M_Chm, chm_ind, impressed_M_waveform[number_of_time_steps*i+time_step]);

		chm_ind = chm_ind + number_of_chm_components[i];
	}
}

void update_impressed_electric_currents_on_gpu(int time_step)
{
	dim3 threads_J, grid_J;
	float *dvE;

	int cej_ind = 0;
	for (int i=0;i<number_of_impressed_J;i++)
	{
		threads_J = dim3(number_of_cej_components[i], 1, 1);
		grid_J    = dim3(1, 1, 1);
		
		if (impressed_J_direction[i]=='x') dvE = dvEx;
		else if (impressed_J_direction[i]=='y')	dvE = dvEy; else dvE = dvEz;

		update_impressed_J_on_gpu<<<grid_J, threads_J>>>
			(dvE, dvJ_indices, dvimpressed_J_Cej, cej_ind, impressed_J_waveform[number_of_time_steps*i+time_step]);

		cej_ind = cej_ind + number_of_cej_components[i];
	}
}

void update_magnetic_fields_for_CPML_on_gpu()
{
	int n_bx = (nxx/TILE_SIZE) + (nxx%TILE_SIZE == 0 ? 0 : 1);
	int n_by = (nyy/TILE_SIZE) + (nyy%TILE_SIZE == 0 ? 0 : 1);
	dim3 threads_cpml = dim3(TILE_SIZE, TILE_SIZE, 1);
	dim3 grid_cpml_xn = dim3(1, n_by, 1);
	dim3 grid_cpml_xp = dim3(2, n_by, 1);
	dim3 grid_cpml_yn = dim3(n_bx, 1, 1);
	dim3 grid_cpml_yp = dim3(n_bx, 2, 1);

	if (is_cpml_xn)
	{
		if (is_TEz)
			update_magnetic_fields_on_gpu_CPML_TEz_xn<<<grid_cpml_xn, threads_cpml>>>(dvPsi_hzx_xn, dvCPsi_hzx_xn, dvHz, dvEy, nxx);

		if (is_TMz)
			update_magnetic_fields_on_gpu_CPML_TMz_xn<<<grid_cpml_xn, threads_cpml>>>(dvPsi_hyx_xn, dvCPsi_hyx_xn, dvHy, dvEz, nxx);
	}

	if (is_cpml_xp)
	{
		if (is_TEz)
			update_magnetic_fields_on_gpu_CPML_TEz_xp<<<grid_cpml_xp, threads_cpml>>>(dvPsi_hzx_xp, dvCPsi_hzx_xp, dvHz, dvEy, nxx, cpml_shift_xp);

		if (is_TMz)
			update_magnetic_fields_on_gpu_CPML_TMz_xp<<<grid_cpml_xp, threads_cpml>>>(dvPsi_hyx_xp, dvCPsi_hyx_xp, dvHy, dvEz, nxx, cpml_shift_xp);
	}

	if (is_cpml_yn)
	{
		if (is_TEz)
			update_magnetic_fields_on_gpu_CPML_TEz_yn<<<grid_cpml_yn, threads_cpml>>>(dvPsi_hzy_yn, dvCPsi_hzy_yn, dvHz, dvEx, nxx);

		if (is_TMz)
			update_magnetic_fields_on_gpu_CPML_TMz_yn<<<grid_cpml_yn, threads_cpml>>>(dvPsi_hxy_yn, dvCPsi_hxy_yn, dvHx, dvEz, nxx);
	}

	if (is_cpml_yp)
	{
		if (is_TEz)
			update_magnetic_fields_on_gpu_CPML_TEz_yp<<<grid_cpml_yp, threads_cpml>>>(dvPsi_hzy_yp, dvCPsi_hzy_yp, dvHz, dvEx, nxx, cpml_shift_yp);

		if (is_TMz)
			update_magnetic_fields_on_gpu_CPML_TMz_yp<<<grid_cpml_yp, threads_cpml>>>(dvPsi_hxy_yp, dvCPsi_hxy_yp, dvHx, dvEz, nxx, cpml_shift_yp);
	}
}

void update_electric_fields_for_CPML_on_gpu()
{
	int n_bx = (nxx/TILE_SIZE) + (nxx%TILE_SIZE == 0 ? 0 : 1);
	int n_by = (nyy/TILE_SIZE) + (nyy%TILE_SIZE == 0 ? 0 : 1);
	dim3 threads_cpml = dim3(TILE_SIZE, TILE_SIZE, 1);
	dim3 grid_cpml_xn = dim3(1, n_by, 1);
	dim3 grid_cpml_xp = dim3(2, n_by, 1);
	dim3 grid_cpml_yn = dim3(n_bx, 1, 1);
	dim3 grid_cpml_yp = dim3(n_bx, 2, 1);

	if (is_cpml_xn)
	{
		if (is_TEz)
			update_electric_fields_on_gpu_CPML_TEz_xn<<<grid_cpml_xn, threads_cpml>>>(dvPsi_eyx_xn, dvCPsi_eyx_xn, dvEy, dvHz, nxx);

		if (is_TMz)
			update_electric_fields_on_gpu_CPML_TMz_xn<<<grid_cpml_xn, threads_cpml>>>(dvPsi_ezx_xn, dvCPsi_ezx_xn, dvEz, dvHy, nxx);
	}

	if (is_cpml_xp)
	{
		if (is_TEz)
			update_electric_fields_on_gpu_CPML_TEz_xp<<<grid_cpml_xp, threads_cpml>>>(dvPsi_eyx_xp, dvCPsi_eyx_xp, dvEy, dvHz, nxx, cpml_shift_xp);

		if (is_TMz)
			update_electric_fields_on_gpu_CPML_TMz_xp<<<grid_cpml_xp, threads_cpml>>>(dvPsi_ezx_xp, dvCPsi_ezx_xp, dvEz, dvHy, nxx, cpml_shift_xp);
	}

	if (is_cpml_yn)
	{
		if (is_TEz)
			update_electric_fields_on_gpu_CPML_TEz_yn<<<grid_cpml_yn, threads_cpml>>>(dvPsi_exy_yn, dvCPsi_exy_yn, dvEx, dvHz, nxx);

		if (is_TMz)
			update_electric_fields_on_gpu_CPML_TMz_yn<<<grid_cpml_yn, threads_cpml>>>(dvPsi_ezy_yn, dvCPsi_ezy_yn, dvEz, dvHx, nxx);
	}

	if (is_cpml_yp)
	{
		if (is_TEz)
			update_electric_fields_on_gpu_CPML_TEz_yp<<<grid_cpml_yp, threads_cpml>>>(dvPsi_exy_yp, dvCPsi_exy_yp, dvEx, dvHz, nxx, cpml_shift_yp);

		if (is_TMz)
			update_electric_fields_on_gpu_CPML_TMz_yp<<<grid_cpml_yp, threads_cpml>>>(dvPsi_ezy_yp, dvCPsi_ezy_yp, dvEz, dvHx, nxx, cpml_shift_yp);

	}
}

bool fdtd_time_marching_loop_on_gpu()
{
	int n_bx = (nxx/TILE_SIZE) + (nxx%TILE_SIZE == 0 ? 0 : 1);
	int n_by = (nyy/TILE_SIZE) + (nyy%TILE_SIZE == 0 ? 0 : 1);
	dim3 threads = dim3(TILE_SIZE, TILE_SIZE, 1);
	dim3 grid = dim3(n_bx, n_by, 1);

	dim3 threads_sef = dim3(number_of_sampled_electric_fields, 1, 1);
	dim3 threads_smf = dim3(number_of_sampled_magnetic_fields, 1, 1);
	dim3 grid_sef = dim3(1, 1, 1);
	dim3 grid_smf = dim3(1, 1, 1);
	
	for (int time_step = 0; time_step<number_of_time_steps;time_step++) 
	{
		if (is_TEz)
			update_magnetic_fields_on_gpu_TEz<<<grid, threads>>>(dvChzh, dvChzex, dvChzey, dvHz, dvEx, dvEy, nxx);

		if (is_TMz)
			update_magnetic_fields_on_gpu_TMz<<<grid, threads>>>(dvChxh, dvChxez, dvChyh, dvChyez, dvHx,  dvHy, dvEz, nxx);

		update_impressed_magnetic_currents_on_gpu(time_step);
		
		update_magnetic_fields_for_CPML_on_gpu();

		capture_sampled_magnetic_fields_on_gpu<<<grid_smf, threads_smf>>>
			(dvHx, dvHy, dvHz, dvsampled_magnetic_fields_component, dvsampled_magnetic_fields_is, 
			dvsampled_magnetic_fields_js, dvsampled_magnetic_fields_sampled_value, time_step, number_of_time_steps, nxx);


		if (is_TEz)
		update_electric_fields_on_gpu_TEz<<<grid, threads>>>(dvCexe, dvCexhz, dvCeye, dvCeyhz,  dvEx, dvEy, dvHz, nxx);

		if (is_TMz)
		update_electric_fields_on_gpu_TMz<<<grid, threads>>>(dvCeze, dvCezhy, dvCezhx, dvHx,  dvHy, dvEz, nxx);
		
		update_impressed_electric_currents_on_gpu(time_step);
		
		update_electric_fields_for_CPML_on_gpu();

		capture_sampled_electric_fields_on_gpu<<<grid_sef, threads_sef>>>
			(dvEz, dvEy, dvEz, dvsampled_electric_fields_component, dvsampled_electric_fields_is, 
			dvsampled_electric_fields_js, dvsampled_electric_fields_sampled_value, time_step, number_of_time_steps, nxx);

		runIterationAndDisplay();
		
		if (time_step%100 == 0)
			printf("timestep: %d \n", time_step);
	}

	return true;
}
bool fdtdIterationOnGpu()
{
	int n_bx = (nxx/TILE_SIZE) + (nxx%TILE_SIZE == 0 ? 0 : 1);
	int n_by = (nyy/TILE_SIZE) + (nyy%TILE_SIZE == 0 ? 0 : 1);
	dim3 threads = dim3(TILE_SIZE, TILE_SIZE, 1);
	dim3 grid = dim3(n_bx, n_by, 1);

	dim3 threads_sef = dim3(number_of_sampled_electric_fields, 1, 1);
	dim3 threads_smf = dim3(number_of_sampled_magnetic_fields, 1, 1);
	dim3 grid_sef = dim3(1, 1, 1);
	dim3 grid_smf = dim3(1, 1, 1);
	
	if (is_TEz)
		update_magnetic_fields_on_gpu_TEz<<<grid, threads>>>(dvChzh, dvChzex, dvChzey, dvHz, dvEx, dvEy, nxx);

	if (is_TMz)
		update_magnetic_fields_on_gpu_TMz<<<grid, threads>>>(dvChxh, dvChxez, dvChyh, dvChyez, dvHx,  dvHy, dvEz, nxx);

	update_impressed_magnetic_currents_on_gpu(time_step);
	
	update_magnetic_fields_for_CPML_on_gpu();

	capture_sampled_magnetic_fields_on_gpu<<<grid_smf, threads_smf>>>
		(dvHx, dvHy, dvHz, dvsampled_magnetic_fields_component, dvsampled_magnetic_fields_is, 
		dvsampled_magnetic_fields_js, dvsampled_magnetic_fields_sampled_value, time_step, number_of_time_steps, nxx);

	if (is_TEz)
	update_electric_fields_on_gpu_TEz<<<grid, threads>>>(dvCexe, dvCexhz, dvCeye, dvCeyhz,  dvEx, dvEy, dvHz, nxx);

	if (is_TMz)
	update_electric_fields_on_gpu_TMz<<<grid, threads>>>(dvCeze, dvCezhy, dvCezhx, dvHx,  dvHy, dvEz, nxx);
	
	update_impressed_electric_currents_on_gpu(time_step);
	
	update_electric_fields_for_CPML_on_gpu();

	capture_sampled_electric_fields_on_gpu<<<grid_sef, threads_sef>>>
		(dvEz, dvEy, dvEz, dvsampled_electric_fields_component, dvsampled_electric_fields_is, 
		dvsampled_electric_fields_js, dvsampled_electric_fields_sampled_value, time_step, number_of_time_steps, nxx);

	if (time_step%100 == 0)
		printf("timestep: %d \n", time_step);
	
	time_step++;

	return true;
}

bool fetchResultsFromGpuMemory()
{
	hipMemcpy(sampled_electric_fields_sampled_value, dvsampled_electric_fields_sampled_value, number_of_sampled_electric_fields*number_of_time_steps*sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(sampled_magnetic_fields_sampled_value, dvsampled_magnetic_fields_sampled_value, number_of_sampled_magnetic_fields*number_of_time_steps*sizeof(float), hipMemcpyDeviceToHost);

	return true;
}

bool deallocateCudaArrays()
{
	hipFree(dvEx);
	hipFree(dvEy);
	hipFree(dvEz);
	hipFree(dvHx);
	hipFree(dvHy);
	hipFree(dvHz);

	hipFree(dvCexe);
	hipFree(dvCexhz);
	hipFree(dvCeye);
	hipFree(dvCeyhz);
	hipFree(dvCeze);
	hipFree(dvCezhy);
	hipFree(dvCezhx);

	hipFree(dvChxh);
	hipFree(dvChxez);
	hipFree(dvChyh);
	hipFree(dvChyez);
	hipFree(dvChzh);
	hipFree(dvChzex);
	hipFree(dvChzey);

	hipFree(dvimpressed_J_Cej);
	hipFree(dvJ_indices);

	hipFree(dvimpressed_M_Chm);
	hipFree(dvM_indices);

	hipFree(dvsampled_electric_fields_component);
	hipFree(dvsampled_electric_fields_is);
	hipFree(dvsampled_electric_fields_js);
	hipFree(dvsampled_electric_fields_sampled_value);

	hipFree(dvsampled_magnetic_fields_component);
	hipFree(dvsampled_magnetic_fields_is);
	hipFree(dvsampled_magnetic_fields_js);
	hipFree(dvsampled_magnetic_fields_sampled_value);

	hipFree(dvminimum_field_value);
	hipFree(dvmaximum_field_value);

	return true;
}