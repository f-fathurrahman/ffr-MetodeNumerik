#pragma once
#include <hip/hip_runtime.h>

#ifndef _fdtd2d_KERNEL_H_
#define _fdtd2d_KERNEL_H_

#define TILE_SIZE 16

__global__ void
update_magnetic_fields_on_gpu_TEz(float* Chzh, float* Chzex, float* Chzey, float* Hz,  float* Ex, float* Ey, int nxx)
{
	__shared__ float sEy[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;

	sEy[ty][tx] = Ey[ci];
	sEy[ty][tx+TILE_SIZE] = Ey[ci+TILE_SIZE];

	__syncthreads();
		Hz[ci] = Chzh[ci] * Hz[ci] + Chzex[ci] * (Ex[ci+nxx]-Ex[ci])  
					+ Chzey[ci] * (sEy[ty][tx+1]-sEy[ty][tx]); 
}

__global__ void
update_magnetic_fields_on_gpu_TMz(float* Chxh, float* Chxez, float* Chyh, float* Chyez, float* Hx,  float* Hy, float* Ez, int nxx)
{
	__shared__ float sEz[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;

	sEz[ty][tx] = Ez[ci];
	sEz[ty][tx+TILE_SIZE] = Ez[ci+TILE_SIZE];

	__syncthreads();

	Hx[ci] = Chxh[ci] * Hx[ci] + Chxez[ci] * (Ez[ci+nxx]-sEz[ty][tx]); 
	Hy[ci] = Chyh[ci] * Hy[ci] + Chyez[ci] * (sEz[ty][tx+1] - sEz[ty][tx]);
}

__global__ void
update_impressed_M_on_gpu(float* H, int* M_indices,  float* M_Chm, int ind, float M_value)
{
	int field_index = M_indices[ind+threadIdx.x];

	H[field_index] = H[field_index] + M_value * M_Chm[ind + threadIdx.x];
}

__global__ void
update_impressed_J_on_gpu(float* E, int* J_indices,  float* J_Cej, int ind, float J_value)
{
	int field_index = J_indices[ind+threadIdx.x];

	E[field_index] = E[field_index] + J_value * J_Cej[ind + threadIdx.x];
}

__global__ void
update_electric_fields_on_gpu_TEz(float* Cexe, float* Cexhz, float* Ceye, float* Ceyhz,  float* Ex, float* Ey, float* Hz, int nxx)
{
	__shared__ float sHz[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	
	sHz[ty][tx] = Hz[ci-TILE_SIZE];
	sHz[ty][tx+TILE_SIZE] = Hz[ci];

	__syncthreads();

	Ex[ci] = Cexe[ci] * Ex[ci] + Cexhz[ci] * (Hz[ci]-Hz[ci-nxx]);
	Ey[ci] = Ceye[ci] * Ey[ci] + Ceyhz[ci] * (sHz[ty][tx+TILE_SIZE]-sHz[ty][tx+TILE_SIZE-1]);
}

__global__ void
update_electric_fields_on_gpu_TMz(float* Ceze, float* Cezhy, float* Cezhx, float* Hx,  float* Hy, float* Ez, int nxx)
{
	__shared__ float sHy[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	
	sHy[ty][tx+TILE_SIZE] = Hy[ci];
	sHy[ty][tx] = Hy[ci-TILE_SIZE];

	__syncthreads();
	Ez[ci] = Ceze[ci] * Ez[ci] + Cezhy[ci] * (sHy[ty][tx+TILE_SIZE]-sHy[ty][tx+TILE_SIZE-1]) + Cezhx[ci] * (Hx[ci]-Hx[ci-nxx]);
}

__global__ void
capture_sampled_electric_fields_on_gpu(float* Ex, float* Ey, float* Ez, char* component, int* is, 
											int* js, float* sampled_value, int time_step, int number_of_time_steps, int nxx)
{
	int tx = threadIdx.x;
	int ci = js[tx]*nxx+is[tx]-1;
	float value;

	switch (component[tx])
	{
	  case 'x' : 
			value = 0.5*(Ex[ci] + Ex[ci-1]);
		break;
	  case 'y' : 
			value = 0.5*(Ey[ci] + Ey[ci-nxx]);
		break;
	  default : 
			value = Ez[ci];
	}
	
	sampled_value[number_of_time_steps*tx+time_step] = value;
}


__global__ void
capture_sampled_magnetic_fields_on_gpu(float* Hx, float* Hy, float* Hz, char* component, int* is, 
											int* js, float* sampled_value, int time_step, int number_of_time_steps, int nxx)
{
	int tx = threadIdx.x;
	int ci = js[tx]*nxx+is[tx]-1;
	float value;

	switch (component[tx])
	{
	  case 'x' : 
			value = 0.5*(Hx[ci] + Hx[ci-nxx]);
		break;
	  case 'y' : 
			value = 0.5*(Hy[ci] + Hy[ci-1]);
		break;
	  default : 
			value = 0.25*(Hz[ci] + Hz[ci-1] + Hz[ci-nxx] + Hz[ci-nxx-1]);
	}
	
	sampled_value[number_of_time_steps*tx+time_step] = value;
}

__constant__ float dvcpml_a_ex_xn[TILE_SIZE];
__constant__ float dvcpml_b_ex_xn[TILE_SIZE];
__constant__ float dvcpml_a_mx_xn[TILE_SIZE];
__constant__ float dvcpml_b_mx_xn[TILE_SIZE];

__constant__ float dvcpml_a_ex_xp[2*TILE_SIZE];
__constant__ float dvcpml_b_ex_xp[2*TILE_SIZE];
__constant__ float dvcpml_a_mx_xp[2*TILE_SIZE];
__constant__ float dvcpml_b_mx_xp[2*TILE_SIZE];

__constant__ float dvcpml_a_ey_yn[TILE_SIZE];
__constant__ float dvcpml_b_ey_yn[TILE_SIZE];
__constant__ float dvcpml_a_my_yn[TILE_SIZE];
__constant__ float dvcpml_b_my_yn[TILE_SIZE];

__constant__ float dvcpml_a_ey_yp[2*TILE_SIZE];
__constant__ float dvcpml_b_ey_yp[2*TILE_SIZE];
__constant__ float dvcpml_a_my_yp[2*TILE_SIZE];
__constant__ float dvcpml_b_my_yp[2*TILE_SIZE];

__global__ void
update_magnetic_fields_on_gpu_CPML_TEz_xn(float* Psi_hzx_xn, float* CPsi_hzx_xn, float* Hz, float* Ey, int nxx)
{
	__shared__ float sEy[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;
	
	int ci = (j+1)*nxx+i;
	int ti = (j+1)*TILE_SIZE+i;

	sEy[ty][tx] = Ey[ci];
	sEy[ty][tx+TILE_SIZE] = Ey[ci+TILE_SIZE];
	__syncthreads();

	Psi_hzx_xn[ti] = dvcpml_b_mx_xn[i] * Psi_hzx_xn[ti] + dvcpml_a_mx_xn[i] * (sEy[ty][tx+1]-sEy[ty][tx]); 	

	Hz[ci] = Hz[ci] + CPsi_hzx_xn[ti] * Psi_hzx_xn[ti];
}

__global__ void
update_magnetic_fields_on_gpu_CPML_TEz_xp(float* Psi_hzx_xp, float* CPsi_hzx_xp, float* Hz, float* Ey, int nxx, int cpml_shift_xp)
{
	__shared__ float sEy[TILE_SIZE][3*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;
	
	int ci = (j+1)*nxx+i+cpml_shift_xp;
	int ti = (j+1)*TILE_SIZE*2+i;

	sEy[ty][tx] = Ey[ci];
	sEy[ty][tx+TILE_SIZE] = Ey[ci+TILE_SIZE];
	__syncthreads();

	Psi_hzx_xp[ti] = dvcpml_b_mx_xp[i] * Psi_hzx_xp[ti] + dvcpml_a_mx_xp[i] * (sEy[ty][tx+1]-sEy[ty][tx]); 	

	Hz[ci] = Hz[ci] + CPsi_hzx_xp[ti] * Psi_hzx_xp[ti];
}

__global__ void
update_magnetic_fields_on_gpu_CPML_TEz_yn(float* Psi_hzy_yn, float* CPsi_hzy_yn, float* Hz, float* Ex, int nxx)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;
	
	int ci = (j+1)*nxx+i;
	int ti = j*nxx+i;

	Psi_hzy_yn[ti] = dvcpml_b_my_yn[j] * Psi_hzy_yn[ti] + dvcpml_a_my_yn[j] * (Ex[ci+nxx] - Ex[ci]); 	

	Hz[ci] = Hz[ci] + CPsi_hzy_yn[ti] * Psi_hzy_yn[ti];
}

__global__ void
update_magnetic_fields_on_gpu_CPML_TEz_yp(float* Psi_hzy_yp, float* CPsi_hzy_yp, float* Hz, float* Ex, int nxx, int cpml_shift_yp)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;
	
	int ci = (j+1+cpml_shift_yp)*nxx+i;
	int ti = j*nxx+i;

	Psi_hzy_yp[ti] = dvcpml_b_my_yp[j] * Psi_hzy_yp[ti] + dvcpml_a_my_yp[j] * (Ex[ci+nxx] - Ex[ci]); 	

	Hz[ci] = Hz[ci] + CPsi_hzy_yp[ti] * Psi_hzy_yp[ti];
}

///////////////////////////////////////////////////////////////////////

__global__ void
update_magnetic_fields_on_gpu_CPML_TMz_xn(float* Psi_hyx_xn, float* CPsi_hyx_xn, float* Hy, float* Ez, int nxx)
{
	__shared__ float sEz[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	int ti = (j+1)*TILE_SIZE+i;

	sEz[ty][tx] = Ez[ci];
	sEz[ty][tx+TILE_SIZE] = Ez[ci+TILE_SIZE];

	__syncthreads();

	Psi_hyx_xn[ti] = dvcpml_b_mx_xn[i] * Psi_hyx_xn[ti] + dvcpml_a_mx_xn[i]*(sEz[ty][tx+1] - sEz[ty][tx]); 

	Hy[ci] = Hy[ci] + CPsi_hyx_xn[ti] * Psi_hyx_xn[ti];
}

__global__ void
update_magnetic_fields_on_gpu_CPML_TMz_xp(float* Psi_hyx_xp, float* CPsi_hyx_xp, float* Hy, float* Ez, int nxx, int cpml_shift_xp)
{
	__shared__ float sEz[TILE_SIZE][3*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i+cpml_shift_xp;
	int ti = (j+1)*TILE_SIZE*2+i;

	sEz[ty][tx] = Ez[ci];
	sEz[ty][tx+TILE_SIZE] = Ez[ci+TILE_SIZE];

	__syncthreads();

	Psi_hyx_xp[ti] = dvcpml_b_mx_xp[i] * Psi_hyx_xp[ti] + dvcpml_a_mx_xp[i]*(sEz[ty][tx+1] - sEz[ty][tx]); 

	Hy[ci] = Hy[ci] + CPsi_hyx_xp[ti] * Psi_hyx_xp[ti];
}

__global__ void
update_magnetic_fields_on_gpu_CPML_TMz_yn(float* Psi_hxy_yn, float* CPsi_hxy_yn, float* Hx, float* Ez, int nxx)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;
	
	int ci = (j+1)*nxx+i;
	int ti = j*nxx+i;

	Psi_hxy_yn[ti] = dvcpml_b_my_yn[j] * Psi_hxy_yn[ti] + dvcpml_a_my_yn[j] * (Ez[ci+nxx] - Ez[ci]); 	

	Hx[ci] = Hx[ci] + CPsi_hxy_yn[ti] * Psi_hxy_yn[ti];
}


__global__ void
update_magnetic_fields_on_gpu_CPML_TMz_yp(float* Psi_hxy_yp, float* CPsi_hxy_yp, float* Hx, float* Ez, int nxx, int cpml_shift_yp)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;
	
	int ci = (j+1+cpml_shift_yp)*nxx+i;
	int ti = j*nxx+i;

	Psi_hxy_yp[ti] = dvcpml_b_my_yp[j] * Psi_hxy_yp[ti] + dvcpml_a_my_yp[j] * (Ez[ci+nxx] - Ez[ci]); 	

	Hx[ci] = Hx[ci] + CPsi_hxy_yp[ti] * Psi_hxy_yp[ti];
}
//------------------------------------------------------------------------

__global__ void
update_electric_fields_on_gpu_CPML_TEz_xn(float* Psi_eyx_xn, float* CPsi_eyx_xn, float* Ey, float* Hz, int nxx)
{
	__shared__ float sHz[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	int ti = (j+1)*TILE_SIZE+i;

	sHz[ty][tx] = Hz[ci-TILE_SIZE];
	sHz[ty][tx+TILE_SIZE] = Hz[ci];

	__syncthreads();

    Psi_eyx_xn[ti] = dvcpml_b_ex_xn[i] * Psi_eyx_xn[ti] + dvcpml_a_ex_xn[i]*(sHz[ty][tx+TILE_SIZE]-sHz[ty][tx+TILE_SIZE-1]); 

	Ey[ci] = Ey[ci] + CPsi_eyx_xn[ti] * Psi_eyx_xn[ti];

}

__global__ void
update_electric_fields_on_gpu_CPML_TEz_xp(float* Psi_eyx_xp, float* CPsi_eyx_xp, float* Ey, float* Hz, int nxx, int cpml_shift_xp)
{
	__shared__ float sHz[TILE_SIZE][3*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i+cpml_shift_xp;
	int ti = (j+1)*TILE_SIZE*2+i;

	sHz[ty][tx] = Hz[ci-TILE_SIZE];
	sHz[ty][tx+TILE_SIZE] = Hz[ci];

	__syncthreads();

    Psi_eyx_xp[ti] = dvcpml_b_ex_xp[i] * Psi_eyx_xp[ti] + dvcpml_a_ex_xp[i]*(sHz[ty][tx+TILE_SIZE]-sHz[ty][tx+TILE_SIZE-1]); 

	Ey[ci] = Ey[ci] + CPsi_eyx_xp[ti] * Psi_eyx_xp[ti];

}

__global__ void
update_electric_fields_on_gpu_CPML_TEz_yn(float* Psi_exy_yn, float* CPsi_exy_yn, float* Ex, float* Hz, int nxx)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	int ti = j*nxx+i;

    Psi_exy_yn[ti] = dvcpml_b_ey_yn[j] * Psi_exy_yn[ti] + dvcpml_a_ey_yn[j]*(Hz[ci]-Hz[ci-nxx]); 

	Ex[ci] = Ex[ci] + CPsi_exy_yn[ti] * Psi_exy_yn[ti];
}

__global__ void
update_electric_fields_on_gpu_CPML_TEz_yp(float* Psi_exy_yp, float* CPsi_exy_yp, float* Ex, float* Hz, int nxx, int cpml_shift_yp)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1+cpml_shift_yp)*nxx+i;
	int ti = j*nxx+i;

    Psi_exy_yp[ti] = dvcpml_b_ey_yp[j] * Psi_exy_yp[ti] + dvcpml_a_ey_yp[j]*(Hz[ci]-Hz[ci-nxx]); 

	Ex[ci] = Ex[ci] + CPsi_exy_yp[ti] * Psi_exy_yp[ti];
}
////////////////////////////////////////////////////////
__global__ void
update_electric_fields_on_gpu_CPML_TMz_xn(float* Psi_ezx_xn, float* CPsi_ezx_xn, float* Ez, float* Hy, int nxx)
{
	__shared__ float sHy[TILE_SIZE][2*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	int ti = (j+1)*TILE_SIZE+i;

	sHy[ty][tx] = Hy[ci-TILE_SIZE];
	sHy[ty][tx+TILE_SIZE] = Hy[ci];

	__syncthreads();

    Psi_ezx_xn[ti] = dvcpml_b_ex_xn[i] * Psi_ezx_xn[ti] + dvcpml_a_ex_xn[i]*(sHy[ty][tx+TILE_SIZE]-sHy[ty][tx+TILE_SIZE-1]); 

	Ez[ci] = Ez[ci] + CPsi_ezx_xn[ti] * Psi_ezx_xn[ti];
}

__global__ void
update_electric_fields_on_gpu_CPML_TMz_xp(float* Psi_ezx_xp, float* CPsi_ezx_xp, float* Ez, float* Hy, int nxx, int cpml_shift_xp)
{
	__shared__ float sHy[TILE_SIZE][3*TILE_SIZE+1];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i+cpml_shift_xp;
	int ti = (j+1)*TILE_SIZE*2+i;

	sHy[ty][tx] = Hy[ci-TILE_SIZE];
	sHy[ty][tx+TILE_SIZE] = Hy[ci];

	__syncthreads();

    Psi_ezx_xp[ti] = dvcpml_b_ex_xp[i] * Psi_ezx_xp[ti] + dvcpml_a_ex_xp[i]*(sHy[ty][tx+TILE_SIZE]-sHy[ty][tx+TILE_SIZE-1]); 

	Ez[ci] = Ez[ci] + CPsi_ezx_xp[ti] * Psi_ezx_xp[ti];

}

__global__ void
update_electric_fields_on_gpu_CPML_TMz_yn(float* Psi_ezy_yn, float* CPsi_ezy_yn, float* Ez, float* Hx, int nxx)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1)*nxx+i;
	int ti = j*nxx+i;

    Psi_ezy_yn[ti] = dvcpml_b_ey_yn[j] * Psi_ezy_yn[ti] + dvcpml_a_ey_yn[j]*(Hx[ci]-Hx[ci-nxx]); 

	Ez[ci] = Ez[ci] + CPsi_ezy_yn[ti] * Psi_ezy_yn[ti];
}

__global__ void
update_electric_fields_on_gpu_CPML_TMz_yp(float* Psi_ezy_yp, float* CPsi_ezy_yp, float* Ez, float* Hx, int nxx, int cpml_shift_yp)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int i = blockIdx.x * blockDim.x + tx;
	int j = blockIdx.y * blockDim.y + ty;

	int ci = (j+1+cpml_shift_yp)*nxx+i;
	int ti = j*nxx+i;

    Psi_ezy_yp[ti] = dvcpml_b_ey_yp[j] * Psi_ezy_yp[ti] + dvcpml_a_ey_yp[j]*(Hx[ci]-Hx[ci-nxx]); 

	Ez[ci] = Ez[ci] + CPsi_ezy_yp[ti] * Psi_ezy_yp[ti];
}

#endif
